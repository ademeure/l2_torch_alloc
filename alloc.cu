#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------------
// [...]
// ---------------------------------------------------------------------------
// Example build & test command:
// nvcc --gpu-architecture=native -Xcompiler -fPIC -shared alloc.cu -o alloc.so -lcuda -lnvrtc && python test.py
// ---------------------------------------------------------------------------
#include <cassert>
#include <cstring>
#include <iostream>
#include <vector>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>
#include <chrono>
#include <fstream>
#include <iterator>
#include <cstdio>
#include <cstdlib>
#include <unistd.h>
#include <string>
#include <ctime>

// ---------------------------------------------------------------------------
// Global configuration variables
// ---------------------------------------------------------------------------
static int g_num_devices = -1; // init on 1st call to get_device_context()
static int g_prealloc_extra_required = 1, g_prealloc_extra_alloc = 10; // configured by set_prealloc_config()
static size_t g_custom_alloc_threshold = 8ULL * 1024ULL * 1024ULL; // use custom allocator above this
static size_t g_free_mapped_start_threshold = 16ULL * 1024ULL * 1024ULL * 1024ULL; // auto unmap on malloc above this
static size_t g_free_mapped_end_threshold   = 2ULL  * 1024ULL * 1024ULL * 1024ULL; // stop auto unmapping at this point
static std::string g_sass_filename; // If empty, don't output SASS. Set via set_output_sass().

// ---------------------------------------------------------------------------
// Compile Time Settings
// ---------------------------------------------------------------------------
//#define DEBUG_PRINTF
constexpr bool   ALWAYS_OUTPUT_SASS = true;    // always output assembly to "sass" file (even if filename is empty)
constexpr int    MAX_SM = 200;                 // enough for all NVIDIA GPUs up to GB300 (but not e.g. MI300X!)
constexpr int    FORCED_HASH = 0;              // 0xB3000 for H100, 0xAB000 for GH200 96GiB, 0x1EF000 for GB200
constexpr int    L2_SIDE_TEST_ITERATIONS = 25; // increases warmup time (at init & per page) but improves accuracy
constexpr bool   TRY_CUDA_FREE_ON_MISS = true; // hipFreeAsync for unknown pointers (e.g. if alloc threshold changes)

constexpr size_t CHUNK_SIZE = 4096;            // 4KiB (= granularity of side switch on H100/GB200)
constexpr size_t PAGE_SIZE  = 2 * 1024 * 1024; // 2MiB (= granularity of NVIDIA MMU pages since Pascal)
constexpr size_t UNBACKED_VIRTUAL_PAGES = 2048ULL * 1024ULL * 1024ULL / PAGE_SIZE; // maximum we allocate in one go

// Offsets into the cpu_side_info array
constexpr int OFFSET_ZEROED_COUNTER    = MAX_SM;
constexpr int OFFSET_AVERAGE_LATENCY   = MAX_SM + 1;
constexpr int OFFSET_SIDE_HASH_MASK    = MAX_SM + 2;
constexpr int OFFSET_NUM_SM_SIDE0      = MAX_SM + 3;
constexpr int OFFSET_NUM_SM_SIDE1      = MAX_SM + 4;
constexpr int OFFSET_MIN_SM_PER_SIDE   = MAX_SM + 5;

// ---------------------------------------------------------------------------
// Everything needed for side-aware page-based allocation
// ---------------------------------------------------------------------------
struct LargeAllocInfo {
    size_t user_requested   = 0;
    void*  base_ptr         = nullptr;  // Points to the correctly aligned section for mapping
    void*  alloc_ptr        = nullptr;  // Original pointer from hipMemAddressReserve
    size_t aligned_size     = 0;
    bool   use_compression  = false;
    hipStream_t last_use_stream = 0;
    std::vector<hipMemGenericAllocationHandle_t> handles; // One handle per 2MiB page
    // Track which side was actually used for each page so we know where to return it:
    std::vector<int> side_used; // side (0 or 1) used for each page (returns to that pool on free)
};

// ---------------------------------------------------------------------------
// Device Context structure to track per-device data
// ---------------------------------------------------------------------------
typedef struct {
    unsigned char side_index[MAX_SM]; // struct to pass this as a single argument to the GPU
} ParamSmSide;

struct DeviceContext {
    // Device properties
    bool initialized = false;
    int device_id = -1;
    int num_sms = 0;

    // CPU memory (used on CPU or passed as kernel parameter)
    int cpu_side_info[MAX_SM * 2];
    ParamSmSide param_sm_side;

    // GPU memory for side info
    unsigned int* gpu_allocator_metadata = nullptr;
    unsigned int* gpu_side_info = nullptr;
    unsigned char* gpu_side_index = nullptr;
    unsigned char* gpu_scratch_buffer = nullptr;

    // Unbacked virtual memory tracking
    unsigned char cpu_page_side[UNBACKED_VIRTUAL_PAGES];

    // Memory pools
    std::vector<hipMemGenericAllocationHandle_t> free_handles_side0;
    std::vector<hipMemGenericAllocationHandle_t> free_handles_side1;
    std::unordered_map<size_t, std::vector<LargeAllocInfo>> large_alloc_cache;
    std::unordered_map<void*, LargeAllocInfo> large_alloc_registry;

    // State tracking
    size_t total_mapped_free = 0;
    bool cross_side_warning_issued = false;
    bool compression_available = false;
    std::unordered_map<size_t, int> size_side_map;

    // Initialize the context for a specific device
    void initialize(hipStream_t stream);

    // -------------------------------------------------------------------
    // NVRTC kernel caching (indexed by header ID)
    // -------------------------------------------------------------------
    struct KernelCacheEntry {
        hipModule_t module = nullptr;                    // compiled module for header
        hipFunction_t funcs[4] = {nullptr,nullptr,nullptr,nullptr};
    };

    std::vector<KernelCacheEntry> kernel_cache;       // index = header ID (0 default)
    std::vector<std::string>      header_strings;     // same index – header source
    std::unordered_map<std::string,int> header_to_id; // reverse lookup
};

// ---------------------------------------------------------------------------
// Helper Functions & Classes
// ---------------------------------------------------------------------------
template<class T>
constexpr inline T ceil_div(T a, T b) { return (a + b - 1) / b; }

template<class... Ts>
__device__ __host__ static void debugf(const char* fmt, Ts... args) {
#ifdef DEBUG_PRINTF
    std::printf(fmt, args...);
#endif
}

#ifndef checkCudaErrors
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

template <typename T>
inline void __checkCudaErrors(T err, const char *file, const int line) {
    if (err != 0) {
        const char *errorStr = "";
        if constexpr (std::is_same<T, hipError_t>::value) { errorStr = hipGetErrorString(err); }
        else if constexpr (std::is_same<T, hipError_t>::value) { hipDrvGetErrorString(err, &errorStr); }
        else if constexpr (std::is_same<T, hiprtcResult>::value) { errorStr = hiprtcGetErrorString(err); }
        fprintf(stderr, "checkCudaErrors() error = %04d \"%s\" from file <%s>, line %d.\n", err, errorStr, file, line);
        assert(false);
    }
}
#endif

// Compiles a CUDA source file (.cu) to CUBIN using NVRTC (asserts on failure)
static void compileFileToCUBIN(hipDevice_t device, char **cubin_out, const char *filename,
                              const char *header_code = nullptr, size_t *cubin_size_out = nullptr,
                              const char *include_path = "/usr/local/cuda/include/") {
    assert(filename && *filename && "NVRTC requires a non-empty filename.");

    std::ifstream file_stream(filename, std::ios::binary);
    assert(file_stream && "Cannot open NVRTC source file.");

    std::string source_code((std::istreambuf_iterator<char>(file_stream)), {});
    std::string final_source = header_code ? (std::string(header_code) + "\n" + source_code) : source_code;

    int major_cc, minor_cc; // this won't include optional 'a' suffix for e.g. H100 sm_90a
    checkCudaErrors(hipDeviceGetAttribute(&major_cc, hipDeviceAttributeComputeCapabilityMajor, device));
    checkCudaErrors(hipDeviceGetAttribute(&minor_cc, hipDeviceAttributeComputeCapabilityMinor, device));
    std::string arch_flag = "--gpu-architecture=sm_" + std::to_string(major_cc) + std::to_string(minor_cc);

    std::vector<std::string> opts_str = {"--generate-line-info", "-use_fast_math",
                                         arch_flag, std::string("-I") + include_path};
    std::vector<const char*> opts_c;
    opts_c.reserve(opts_str.size());
    for(const auto& s : opts_str) opts_c.push_back(s.c_str());

    hiprtcProgram program;
    checkCudaErrors(hiprtcCreateProgram(&program, final_source.c_str(), filename, 0, nullptr, nullptr));
    hiprtcResult compile_res = hiprtcCompileProgram(program, opts_c.size(), opts_c.data());

    size_t log_size = 0;
    checkCudaErrors(hiprtcGetProgramLogSize(program, &log_size));
    if (log_size > 1) { // Print log even on success (for warnings)
        std::string log(log_size, '\0');
        checkCudaErrors(hiprtcGetProgramLog(program, &log[0]));
        std::cerr << "NVRTC Log (" << filename << "):\n" << log << std::endl;
    }

    if (compile_res != HIPRTC_SUCCESS) {
        const char* error_string = hiprtcGetErrorString(compile_res);
        std::cerr << "Error: NVRTC compilation failed for '" << filename
                  << "' with error: " << (error_string ? error_string : "Unknown NVRTC error")
                  << " (Code: " << compile_res << "). Check NVRTC log above for details.\n";
        checkCudaErrors(hiprtcDestroyProgram(&program));
        assert(false);
    }

    size_t cubin_size = 0;
    checkCudaErrors(hiprtcGetBitcodeSize(program, &cubin_size));
    if (cubin_size_out) *cubin_size_out = cubin_size;

    *cubin_out = static_cast<char*>(malloc(cubin_size));
    assert(*cubin_out && "Failed malloc for CUBIN");

    checkCudaErrors(hiprtcGetBitcode(program, *cubin_out));
    checkCudaErrors(hiprtcDestroyProgram(&program));

    // Extract SASS from the generated CUBIN and append it to the output file
    if (ALWAYS_OUTPUT_SASS || !g_sass_filename.empty()) {
        char tmpPath[] = "tmp_cubin";
        FILE* tmpFile = fopen(tmpPath, "wb");
        if (tmpFile) {
            fwrite(*cubin_out, 1, cubin_size, tmpFile);
            fclose(tmpFile);

            std::string cmd = "cuobjdump --dump-sass " + std::string(tmpPath) + " 2>/dev/null";
            if (FILE* pipe = popen(cmd.c_str(), "r")) {
                std::ofstream sass_file(g_sass_filename.empty() ? "sass" : g_sass_filename, std::ios::app);
                if (sass_file.is_open()) {
                    char buffer[4096];
                    auto now = std::chrono::system_clock::now();
                    std::time_t now_c = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
                    std::strftime(buffer, sizeof(buffer), "%F %T", std::localtime(&now_c));

                    sass_file << "==================== Start SASS dump (" << buffer << ") =====================\n";
                    while (fgets(buffer, sizeof(buffer), pipe)) {
                        sass_file << buffer;
                    }
                    sass_file << "===================== End SASS dump ======================\n";
                }
                pclose(pipe);
            }
        }
        remove(tmpPath);
    }
}

static hipModule_t loadCUBIN(char *cubin, hipDevice_t cuDevice, bool free_cubin=true) {
    hipModule_t module;
    checkCudaErrors(hipModuleLoadData(&module, cubin));
    if (free_cubin) free(cubin);
    return module;
}

// Multi-GPU only: save current device and restore it when destroyed (out of scope)
class ScopedSetDevice {
public:
    explicit ScopedSetDevice(int new_device) {
        if (g_num_devices != 1) {
            hipGetDevice(&old_device);
            hipSetDevice(new_device);
        }
    }
    ~ScopedSetDevice() {
        if (g_num_devices != 1) {
            hipSetDevice(old_device);
        }
    }
private:
    int old_device;
};

// ---------------------------------------------------------------------------
// Device functions & kernels for side info
// ---------------------------------------------------------------------------
__device__ __forceinline__ int test_latency_l2(unsigned int* data, size_t offset) {
    unsigned int old_value = atomicExch(&data[offset], 0); // also warms up the cache!
    long long int start_clock = clock64();
    for (int i = 0; i < L2_SIDE_TEST_ITERATIONS; i++) {
        int value = atomicInc(&data[offset], 99999999);
        offset += (value > L2_SIDE_TEST_ITERATIONS*10) ? 1 : 0;
    }
    int latency = clock64() - start_clock;
    data[offset] = old_value;
    return latency;
}

__global__ void init_side_info(unsigned int* base_page, unsigned int *side_info, unsigned char *side_index) {
    if (threadIdx.x == 0) {
        int smid;
        asm volatile("mov.u32 %0, %smid;\n" : "=r"(smid) :);

        int offset = 4 * smid;
        assert(offset * sizeof(int) < CHUNK_SIZE);

        __nanosleep(((smid+4)% 16) * 2000 + 100);
        int total_latency = test_latency_l2(base_page, offset);
        side_info[smid] = total_latency;
        atomicAdd(&side_info[OFFSET_AVERAGE_LATENCY], total_latency);

        int num_done = atomicInc(&side_info[OFFSET_ZEROED_COUNTER], gridDim.x - 1);
        if (num_done == gridDim.x - 1) {
            int average_latency = side_info[OFFSET_AVERAGE_LATENCY] / gridDim.x;
            debugf("Average L2-latency threshold: %.1f\n", (float)average_latency / (float)L2_SIDE_TEST_ITERATIONS);

            // SM0 is always side 0 (everything else is relative to it)
            int far_side =  (side_info[0] > average_latency) ? 0 : 1;
            int near_side = (side_info[0] > average_latency) ? 1 : 0;

            int side0_counter = 0, side1_counter = 0;
            for (int i = 0; i < gridDim.x; i++) {
                int latency = side_info[i];
                side_info[i] = (latency > average_latency) ? far_side : near_side;
                if ((side_info[i] & 1) == 0) {
                    side_info[i] |= (side0_counter++) << 1;
                } else {
                    side_info[i] |= (side1_counter++) << 1;
                }
                side_index[i] = (unsigned char)side_info[i];
                debugf("[SM %3d] L2-latency = %.1f -> side=%d idx=%d\n",
                       i, (float)latency / (float)L2_SIDE_TEST_ITERATIONS,
                       (side_info[i] & 1), (side_info[i] >> 1));
            }
            side_info[OFFSET_AVERAGE_LATENCY] = average_latency;
            side_info[OFFSET_NUM_SM_SIDE0]    = side0_counter;
            side_info[OFFSET_NUM_SM_SIDE1]    = side1_counter;
            side_info[OFFSET_MIN_SM_PER_SIDE] = min(side0_counter, side1_counter);

            if constexpr (FORCED_HASH == 0) {
                unsigned long long int addr_int = reinterpret_cast<unsigned long long int>(base_page);
                if (addr_int % ((size_t)PAGE_SIZE) != 0) {
                    debugf("ERROR: base_page not 2MiB-aligned\n");
                    assert(false);
                    return;
                }
                int base_side = side_info[smid] & 1;
                int check_start_bit = 4;
                int check_last_bit  = 20;
                int toggle_bits = 0;
                for (int i = check_start_bit; i <= check_last_bit; i++) {
                    int bitmask = 1 << i;
                    int offset2 = bitmask / sizeof(int);
                    int total_latency2 = test_latency_l2(base_page, offset2);
                    int offset_side = (total_latency2 > average_latency)? far_side : near_side;
                    if (offset_side != base_side) {
                        toggle_bits |= bitmask;
                    }
                }
                side_info[OFFSET_SIDE_HASH_MASK] = toggle_bits;
                debugf("Detected side-hash bits: 0x%X\n", toggle_bits);
                if (!(toggle_bits & CHUNK_SIZE) || (toggle_bits & (CHUNK_SIZE - 1))) {
                    printf("\nERROR: CHUNK_SIZE %d doesn't work with hash %x\n\n", (int)CHUNK_SIZE, toggle_bits);
                    assert(false);
                }
            } else {
                side_info[OFFSET_SIDE_HASH_MASK] = FORCED_HASH;
            }
        }
    } else if (threadIdx.x >= 32) {
        __nanosleep(10000);
    }
}

__global__ void test_page_latency(unsigned int* ptr, unsigned int *side_info, unsigned char* page_side, int num_pages)
{
    if (threadIdx.x == 0) {
        int smid;
        asm volatile("mov.u32 %0, %smid;\n" : "=r"(smid) :);
        int near_side = side_info[smid] & 1;
        int far_side  = 1 - near_side;
        int average_latency = side_info[OFFSET_AVERAGE_LATENCY];

        debugf("Testing %u pages.\n", num_pages);
        for (int i = 0; i < num_pages; i++) {
            size_t offset = (size_t)i * (PAGE_SIZE / sizeof(unsigned int));
            int total_latency = test_latency_l2(ptr, offset);
            page_side[i] = (total_latency > average_latency) ? far_side : near_side;
            debugf("[SM %3d] Page %3d: L2-latency = %.1f (raw: %u) -> side=%d\n",
                   smid, i, (float)total_latency / (float)L2_SIDE_TEST_ITERATIONS, total_latency, page_side[i]);
        }
    }
}

// Device contexts array - indexed by device ID
static std::vector<DeviceContext> g_deviceContexts;

static DeviceContext& get_device_context(int device=-1) {
    if (g_num_devices < 0) {
        hipGetDeviceCount(&g_num_devices);
        g_deviceContexts.resize(g_num_devices);
    }

    if (device < 0) {
        if (g_num_devices == 1) {
            device = 0;
        } else {
            hipGetDevice(&device);
        }
    }

    if (device >= g_deviceContexts.size()) {
        assert(device < g_num_devices);
        g_deviceContexts.resize(g_num_devices);
    }

    DeviceContext& ctx = g_deviceContexts[device];
    if (!ctx.initialized) {
        ctx.device_id = device;
        ctx.initialize(0);
    }
    return ctx;
}

// NVRTC kernel compilation helper (for side_aware_memcpy)
// TODO: this isn't actually memcpy anymore
static hipFunction_t getMemcpyKernel(DeviceContext &ctx, int header_id, bool use_64bit) {
    assert(header_id < ctx.kernel_cache.size() && header_id < ctx.header_strings.size());
    DeviceContext::KernelCacheEntry &entry = ctx.kernel_cache[header_id];

    int idx = use_64bit ? 1 : 0; // select variant
    if (entry.funcs[idx] == nullptr) {
        assert(entry.module == nullptr);

        hipDevice_t cuDevice;
        hipCtxGetDevice(&cuDevice);

        char *cubin = nullptr;
        const std::string& s = ctx.header_strings[header_id];
        const char* header_ptr = s.empty() ? nullptr : s.c_str();
        compileFileToCUBIN(cuDevice, &cubin, "sideaware_kernels.cu", header_ptr);
        entry.module = loadCUBIN(cubin, cuDevice);

        const char* fn_names[2] = { "side_aware_memcpy_32", "side_aware_memcpy_64" };
        for (int i = 0; i < 2; i++) {
            hipFunction_t ftmp;
            hipError_t rc = hipModuleGetFunction(&ftmp, entry.module, fn_names[i]);
            if (rc != hipSuccess) {
                const char *errStr = nullptr; hipDrvGetErrorString(rc,&errStr);
                std::cerr << "Failed to get " << fn_names[i] << " : " << (errStr?errStr:"") << std::endl;
                std::abort();
            }
            entry.funcs[i] = ftmp;
        }
    }
    return entry.funcs[idx];
}

// Query the device allocation constraints
static void init_allocation_constraints(DeviceContext& ctx)
{
    int comp_available;
    hipDeviceGetAttribute(&comp_available, CU_DEVICE_ATTRIBUTE_GENERIC_COMPRESSION_SUPPORTED, ctx.device_id);
    ctx.compression_available = (comp_available != 0);

    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = ctx.device_id;

    size_t granularity;
    hipError_t res = hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);
    assert(res == hipSuccess && granularity == PAGE_SIZE); // Verify granularity equals PAGE_SIZE

    if (ctx.compression_available) {
        prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;
        res = hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);
        assert(res == hipSuccess && granularity == PAGE_SIZE);
    }
}

static hipMemAllocationProp get_allocation_constraints(DeviceContext& ctx, bool use_compression=false)
{
    use_compression &= ctx.compression_available;

    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = ctx.device_id;
    prop.allocFlags.compressionType = (use_compression) ? CU_MEM_ALLOCATION_COMP_GENERIC : 0;
    return prop;
}

void DeviceContext::initialize(hipStream_t stream) {
    if (initialized) return;
    ScopedSetDevice guard(device_id);

    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, device_id);

    init_allocation_constraints(*this);
    hipMalloc(&gpu_allocator_metadata, PAGE_SIZE);
    assert((uintptr_t)gpu_allocator_metadata % PAGE_SIZE == 0);
    gpu_side_info = &gpu_allocator_metadata[16 * 1024];
    gpu_side_index = (unsigned char*)&gpu_allocator_metadata[32 * 1024];
    gpu_scratch_buffer = (unsigned char*)&gpu_allocator_metadata[128 * 1024];

    init_side_info<<<num_sms, 512, 0, stream>>>(gpu_allocator_metadata, gpu_side_info, gpu_side_index);

    unsigned char cpu_side_index[MAX_SM];
    hipMemcpyAsync(cpu_side_index, gpu_side_index, MAX_SM * sizeof(unsigned char), hipMemcpyDeviceToHost, stream);
    hipMemcpy(cpu_side_info, gpu_side_info, MAX_SM * 2 * sizeof(unsigned int), hipMemcpyDeviceToHost); // syncs both

    for (int i = 0; i < MAX_SM; i++) {
        param_sm_side.side_index[i] = cpu_side_index[i];
    }
    header_strings.resize(1);
    kernel_cache.resize(1);

    initialized = true;
}

// ---------------------------------------------------------------------------
// Unmap free blocks if threshold is exceeded
// ---------------------------------------------------------------------------
static void unmapFreeLargeAllocations(DeviceContext& ctx, size_t start_threshold=0, size_t end_threshold=0) {
    if (ctx.total_mapped_free <= start_threshold) return;

    ScopedSetDevice guard(ctx.device_id);
    hipDeviceSynchronize();

    for (auto &kv : ctx.large_alloc_cache) {
        auto &vec = kv.second;
        auto it = vec.begin();

        while (it != vec.end() && ctx.total_mapped_free > end_threshold) {
            LargeAllocInfo &blk = *it;
            size_t size = blk.aligned_size;
            size_t num_pages = blk.aligned_size / PAGE_SIZE;
            hipDeviceptr_t base = (hipDeviceptr_t)blk.base_ptr;

            for (size_t i = 0; i < num_pages; i++) {
                hipMemUnmap(base + i * PAGE_SIZE, PAGE_SIZE);
                // Return handle to the side it was allocated from:
                int side = blk.side_used[i];
                if (side == 0) {
                    ctx.free_handles_side0.push_back(blk.handles[i]);
                } else {
                    ctx.free_handles_side1.push_back(blk.handles[i]);
                }
            }
            // Use alloc_ptr for freeing address space, not base_ptr
            hipMemAddressFree((hipDeviceptr_t)blk.alloc_ptr, blk.aligned_size + 2 * PAGE_SIZE);

            assert(ctx.total_mapped_free >= size);
            ctx.total_mapped_free -= size;
            it = vec.erase(it);
        }
        if (ctx.total_mapped_free <= end_threshold)
            break;
    }
    hipDeviceSynchronize();
}

static size_t release_unused_memory_device(int device) {
    DeviceContext& ctx = get_device_context(device);
    ScopedSetDevice guard(device);

    // Unmap all cached blocks then release handles from side0/side1 pools
    unmapFreeLargeAllocations(ctx);

    size_t freed_memory = 0;
    for (auto &h : ctx.free_handles_side0) {
        hipMemRelease(h);
        freed_memory += PAGE_SIZE;
    }
    ctx.free_handles_side0.clear();

    for (auto &h : ctx.free_handles_side1) {
        hipMemRelease(h);
        freed_memory += PAGE_SIZE;
    }
    ctx.free_handles_side1.clear();

    hipDeviceSynchronize();
    return freed_memory;
}

// ---------------------------------------------------------------------------
// Pre-allocate new physical handles (=2MiB pages of GPU physical memory)
// Called when we run out but allocates some extra to reduce future calls
// This is useful because this requires fully synchronous operations
// ---------------------------------------------------------------------------
static hipError_t preAllocateHandles(DeviceContext& ctx, int countNeeded, bool useCompression, hipStream_t stream)
{
    if (countNeeded == 0) return hipSuccess;

    hipMemAllocationProp prop = get_allocation_constraints(ctx, useCompression);
    hipError_t lastError = hipSuccess;
    int totalAllocated = 0;

    // We'll do it in batches so we don't exceed UNBACKED_VIRTUAL_PAGES at once.
    while (countNeeded > 0) {
        int batch = std::min<int>(countNeeded, UNBACKED_VIRTUAL_PAGES);
        int batchAllocated = 0;

        // Reserve a *temporary* VA space for up to 'batch' pages (freed before returning)
        hipDeviceptr_t dptr;
        lastError = hipMemAddressReserve(&dptr, batch * PAGE_SIZE, 0, 0, 0);
        if (lastError != hipSuccess) {
            break; // Can't continue this batch, but process what we have so far
        }

        // Create + map each handle:
        std::vector<hipMemGenericAllocationHandle_t> tempHandles(batch);
        for (int i = 0; i < batch; i++) {
            lastError = hipMemCreate(&tempHandles[i], PAGE_SIZE, &prop, 0);
            if (lastError != hipSuccess) {
                tempHandles.resize(i);
                batch = i; // adjust batch size
                break;
            }

            lastError = hipMemMap(dptr + i * PAGE_SIZE, PAGE_SIZE, 0, tempHandles[i], 0);
            if (lastError != hipSuccess) {
                hipMemRelease(tempHandles[i]);
                tempHandles.resize(i);
                batch = i;
                break;
            }
            batchAllocated++;
        }

        // If failed to allocate anything in this iteration, free the VA space and continue
        if (batchAllocated == 0) {
            hipMemAddressFree(dptr, batch * PAGE_SIZE);
            break;
        }

        // Set read/write access for everything we allocated (in the VA space)
        hipMemAccessDesc accessDesc;
        accessDesc.location.id   = prop.location.id;
        accessDesc.location.type = prop.location.type;
        accessDesc.flags         = hipMemAccessFlagsProtReadWrite;
        lastError = hipMemSetAccess(dptr, batchAllocated * PAGE_SIZE, &accessDesc, 1);
        if (lastError != hipSuccess) {
            for (int i = 0; i < batchAllocated; i++) {
                hipMemUnmap(dptr + i * PAGE_SIZE, PAGE_SIZE);
                hipMemRelease(tempHandles[i]);
            }
            hipMemAddressFree(dptr, batch * PAGE_SIZE);
            break;
        }

        // Zero out memory before using it (TODO: do we need this, or is it overkill?)
        hipDeviceSynchronize();
        lastError = hipMemsetD8(dptr, 0, batchAllocated * PAGE_SIZE);
        if (lastError != hipSuccess) {
            printf("ERROR: Failed to zero out memory before test (this should never happen?!)\n");
            assert(false);
            return lastError;
        }

        // Classify each page (side 0 or 1) by calling test_page_latency
        test_page_latency<<<1, 512, 0, stream>>>(
            reinterpret_cast<unsigned int*>(dptr), ctx.gpu_side_info, ctx.gpu_scratch_buffer, batchAllocated);
        hipMemcpy(ctx.cpu_page_side, ctx.gpu_scratch_buffer, batchAllocated, hipMemcpyDeviceToHost);

        // Unmap + place each handle in the correct side's pool
        for (int i = 0; i < batchAllocated; i++) {
            hipMemUnmap(dptr + i * PAGE_SIZE, PAGE_SIZE);
            int side = ctx.cpu_page_side[i];
            if (side == 0) {
                ctx.free_handles_side0.push_back(tempHandles[i]);
            } else {
                ctx.free_handles_side1.push_back(tempHandles[i]);
            }
        }
        hipMemAddressFree(dptr, batch * PAGE_SIZE);

        totalAllocated += batchAllocated;
        countNeeded -= batchAllocated;

        if (batchAllocated < batch) {
            break; // probably out of GPU memory?
        }
    }

    // If we allocated anything at all, consider it at least a partial success
    if (totalAllocated > 0) {
        return hipSuccess;
    }
    return lastError;
}

static hipError_t ensureFreeHandlesAvailable(DeviceContext& ctx, size_t needed, bool useCompression, hipStream_t stream)
{
    size_t needed_per_side = (needed + g_prealloc_extra_required) / 2;
    size_t free0 = ctx.free_handles_side0.size();
    size_t free1 = ctx.free_handles_side1.size();
    size_t needed_side0 = (needed_per_side > free0) ? (needed_per_side - free0) : 0;
    size_t needed_side1 = (needed_per_side > free1) ? (needed_per_side - free1) : 0;
    size_t needed_worst_case = max(needed_side0, needed_side1);
    size_t needed_both_sides = 2 * needed_worst_case;

    if (needed_both_sides > 0) {
        needed_both_sides += g_prealloc_extra_alloc;
        hipError_t rc = preAllocateHandles(ctx, needed_both_sides, useCompression, stream);
        return rc;
    }
    return hipSuccess;
}

constexpr inline int pickSideFromVA(uint64_t va)
{
    return (int)((va >> 21) & 1ULL);
}

static hipError_t allocateCompressible(DeviceContext& ctx, LargeAllocInfo &info, size_t size, bool use_compression=false)
{
    info.aligned_size = ((size + PAGE_SIZE - 1) / PAGE_SIZE) * PAGE_SIZE;
    info.use_compression = (use_compression && ctx.compression_available);

    // 1) Pre-check we have enough free handles, otherwise allocate more:
    size_t num_pages = info.aligned_size / PAGE_SIZE;
    hipError_t rc = ensureFreeHandlesAvailable(ctx, num_pages, info.use_compression, info.last_use_stream);
    if (rc != hipSuccess) {
        return rc;
    }

    // Determine the desired "start side" for this allocation size
    // If we haven't seen this size before, choose the side with fewer handles
    if (ctx.size_side_map.find(info.aligned_size) == ctx.size_side_map.end()) {
        int side0_free = ctx.free_handles_side0.size();
        int side1_free = ctx.free_handles_side1.size();
        ctx.size_side_map[info.aligned_size] = (side0_free >= side1_free) ? 0 : 1;
    }
    int desiredStartSide = ctx.size_side_map[info.aligned_size];

    // 2) Reserve VA space with EXTRA space (extra page = 2MiB)
    // This ensures we'll find a properly-aligned section with the desired side
    hipDeviceptr_t allocPtr = 0;
    size_t extraSize = info.aligned_size + PAGE_SIZE;
    rc = hipMemAddressReserve(&allocPtr, extraSize, 0, 0, 0);
    if (rc != hipSuccess) {
        return rc;
    }

    info.alloc_ptr = reinterpret_cast<void*>(allocPtr);
    hipDeviceptr_t basePtr = allocPtr;

    // If the first page is not on the desired side, try the second page
    int firstPageSide = pickSideFromVA((uint64_t)basePtr);
    if (firstPageSide != desiredStartSide) {
        basePtr += PAGE_SIZE; // Skip to next 2MiB page
    }

    // Verify that this page has the desired side
    int basePageSide = pickSideFromVA((uint64_t)basePtr);
    if (basePageSide != desiredStartSide) {
        // This should never happen with our hash function using bit 21?!
        printf("ERROR: Failed to find desired side in 2 consecutive 2MiB pages\n");
        hipMemAddressFree(allocPtr, extraSize);
        return hipErrorUnknown;
    }

    // Set the basePtr to point to the section with the desired side
    info.base_ptr = reinterpret_cast<void*>(basePtr);
    info.handles.resize(num_pages);
    info.side_used.resize(num_pages);

    // 3) Map each page from whichever side is indicated by bit 21
    for (size_t i = 0; i < num_pages; i++) {
        uint64_t thisVa = (uint64_t)(basePtr + i * PAGE_SIZE);
        int desiredSide = pickSideFromVA(thisVa);

        // pop from the correct side if available
        std::vector<hipMemGenericAllocationHandle_t>* correctPool =
            (desiredSide == 0) ? &ctx.free_handles_side0 : &ctx.free_handles_side1;
        std::vector<hipMemGenericAllocationHandle_t>* otherPool   =
            (desiredSide == 0) ? &ctx.free_handles_side1 : &ctx.free_handles_side0;

        if (!correctPool->empty()) {
            info.handles[i] = correctPool->back();
            correctPool->pop_back();
            info.side_used[i] = desiredSide;
        } else {
            // Try to fallback to the other side
            if (otherPool->empty()) {
                // We've run out of GPU memory on both sides
                hipMemAddressFree(allocPtr, extraSize);
                return hipErrorOutOfMemory;
            }
            // Warn the first time this happens
            if (!ctx.cross_side_warning_issued) {
                printf("WARNING: Cross-side handle usage on device %d!\n", ctx.device_id);
                ctx.cross_side_warning_issued = true;
            }
            info.handles[i] = otherPool->back();
            otherPool->pop_back();
            info.side_used[i] = desiredSide;
        }

        rc = hipMemMap(basePtr + i * PAGE_SIZE, PAGE_SIZE, 0, info.handles[i], 0);
        if (rc != hipSuccess) {
            // Return the handle to whichever side we took it from
            if (info.side_used[i] == 0) {
                ctx.free_handles_side0.push_back(info.handles[i]);
            } else {
                ctx.free_handles_side1.push_back(info.handles[i]);
            }
            // Cleanup on allocation failure
            for (size_t j = 0; j < i; j++) {
                hipMemUnmap(basePtr + j * PAGE_SIZE, PAGE_SIZE);
                if (info.side_used[j] == 0) {
                    ctx.free_handles_side0.push_back(info.handles[j]);
                } else {
                    ctx.free_handles_side1.push_back(info.handles[j]);
                }
            }
            hipMemAddressFree(allocPtr, extraSize);
            return rc;
        }
    }

    // 4) Enable read/write access for the entire allocation in the VA space
    hipMemAccessDesc accessDesc;
    accessDesc.location.id   = ctx.device_id;
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.flags         = hipMemAccessFlagsProtReadWrite;
    rc = hipMemSetAccess(basePtr, info.aligned_size, &accessDesc, 1);
    if (rc != hipSuccess) {
        // Cleanup on allocation failure
        for (size_t i = 0; i < num_pages; i++) {
            hipMemUnmap(basePtr + i * PAGE_SIZE, PAGE_SIZE);
            if (info.side_used[i] == 0) ctx.free_handles_side0.push_back(info.handles[i]);
            else                       ctx.free_handles_side1.push_back(info.handles[i]);
        }
        hipMemAddressFree(allocPtr, extraSize);
        return rc;
    }

    // 5) Zero memory to be safe (TODO: make this configurable)
    hipDeviceSynchronize();
    rc = hipMemsetD8(basePtr, 0, info.aligned_size);
    if (rc != hipSuccess) {
        // Cleanup on allocation failure
        for (size_t i = 0; i < num_pages; i++) {
            hipMemUnmap(basePtr + i * PAGE_SIZE, PAGE_SIZE);
            if (info.side_used[i] == 0) ctx.free_handles_side0.push_back(info.handles[i]);
            else                       ctx.free_handles_side1.push_back(info.handles[i]);
        }
        hipMemAddressFree(allocPtr, extraSize);
        return rc;
    }

    return hipSuccess;
}

static void* sideaware_reuse_alloc(DeviceContext& ctx, size_t alignedSize, hipStream_t currentStream)
{
    auto &vec = ctx.large_alloc_cache[alignedSize];
    if (vec.empty()) return nullptr;

    LargeAllocInfo info = vec.back();
    assert(ctx.total_mapped_free >= info.aligned_size);
    ctx.total_mapped_free -= info.aligned_size;
    vec.pop_back();

    // If it's a different stream, we might need to synchronize
    // TODO: is this safe or do we need a full device synchronization?
    if (info.last_use_stream != currentStream) {
        hipStreamSynchronize(info.last_use_stream);
        info.last_use_stream = currentStream;
    }

    // Insert back into active registry
    ctx.large_alloc_registry[info.base_ptr] = info;
    return info.base_ptr;
}

static void* sideaware_new_alloc(DeviceContext& ctx, size_t userSize, hipStream_t stream)
{
    LargeAllocInfo info;
    info.user_requested = userSize;
    info.last_use_stream = stream;
    hipError_t rc = allocateCompressible(ctx, info, userSize);
    if (rc != hipSuccess)
        return nullptr;
    ctx.large_alloc_registry[info.base_ptr] = info;
    return info.base_ptr;
}

static void* sideaware_malloc_large(DeviceContext& ctx, size_t size, hipStream_t stream)
{
    size_t alignedSize = ((size + PAGE_SIZE - 1) / PAGE_SIZE) * PAGE_SIZE;

    // 1) Try exact-size reuse
    void* p = sideaware_reuse_alloc(ctx, alignedSize, stream);
    if (p) return p;

    // 2) Unmap free pages if over threshold & try again
    unmapFreeLargeAllocations(ctx, g_free_mapped_start_threshold, g_free_mapped_end_threshold);
    p = sideaware_new_alloc(ctx, size, stream);
    if (p) return p;

    // 3) Try fully releasing all unused memory & try again
    release_unused_memory_device(ctx.device_id);
    return sideaware_new_alloc(ctx, size, stream);
}

static void sideaware_free_large(DeviceContext& ctx, void* ptr, hipStream_t stream)
{
    if (!ptr) return;
    auto it = ctx.large_alloc_registry.find(ptr);
    if (it == ctx.large_alloc_registry.end()) {
        if constexpr (TRY_CUDA_FREE_ON_MISS) {
            hipFreeAsync(ptr, stream);
        } else {
            printf("ERROR: Failed to find large allocation to free on device %d\n", ctx.device_id);
            assert(false);
        }
        return;
    }

    LargeAllocInfo info = it->second;
    ctx.large_alloc_registry.erase(it);
    ctx.large_alloc_cache[info.aligned_size].push_back(info);
    ctx.total_mapped_free += info.aligned_size;
}

// ---------------------------------------------------------------------------
// PUBLIC API
// ---------------------------------------------------------------------------
extern "C" {

void* sideaware_malloc(size_t size, int device, hipStream_t stream) {
    debugf("sideaware_malloc(%zu, %d, %p)\n", size, device, stream);
    void* p = nullptr;
    ScopedSetDevice guard(device);
    DeviceContext& ctx = get_device_context(device);

    if (size >= g_custom_alloc_threshold) {
        p = sideaware_malloc_large(ctx, size, stream);
    } else {
        hipError_t err = hipMallocAsync(&p, size, stream);
        if (err == hipSuccess) {
            return p;
        }

        unmapFreeLargeAllocations(ctx);
        err = hipMallocAsync(&p, size, stream);
        if (err != hipSuccess) {
            p = nullptr;
        }
    }

    return p;
}

void sideaware_free(void* ptr, size_t size, int device, hipStream_t stream) {
    debugf("sideaware_free(%p, %zu, %d, %p)\n", ptr, size, device, stream);

    if (!ptr) return;
    DeviceContext& ctx = get_device_context(device);
    ScopedSetDevice guard(device);

    if (size >= g_custom_alloc_threshold) {
        sideaware_free_large(ctx, ptr, stream);
    } else {
        hipFreeAsync(ptr, stream);
    }
}

size_t sideaware_release_unused() {
    size_t total_freed = 0;
    for (size_t i = 0; i < g_deviceContexts.size(); i++) {
        if (g_deviceContexts[i].initialized) {
            total_freed += release_unused_memory_device(i);
        }
    }
    return total_freed;
}

// ---------------------------------------------------------------------------
// Generic element‑wise kernel launcher. header_id selects runtime‑compiled
// module to use (0 = default memcpy).
// TODO: assumes device == current device, is this always true with PyTorch?
// ---------------------------------------------------------------------------
void sideaware_elementwise(void* dst, const void* src, size_t size, int device, hipStream_t stream, int header_id) {
    if (size == 0 || dst == nullptr || src == nullptr) return;
    DeviceContext& ctx = get_device_context(device);
    assert(header_id >= 0 && header_id < ctx.kernel_cache.size());

    int sm_per_side = ctx.cpu_side_info[OFFSET_MIN_SM_PER_SIDE];
    int hash = ctx.cpu_side_info[OFFSET_SIDE_HASH_MASK] | (1 << 21);

    unsigned int byte_start = (unsigned long long)src & 15;
    unsigned int byte_end = ((unsigned long long)src + size) & 15;
    size_t size_aligned = size;

    if (byte_start) {
        src = (const uint4*)(((const unsigned char*)src) + sizeof(uint4) - byte_start);
        dst = (uint4*)(((unsigned char*)dst) + sizeof(uint4) - byte_start);
        size_aligned -= (sizeof(uint4) - byte_start);
    }
    if (byte_end) {
        size_aligned -= byte_end;
    }

    unsigned int size_aligned_32b = (unsigned int)size_aligned;
    uint4* dst4 = (uint4*)dst;
    const uint4* src4 = (const uint4*)src;


    bool use64 = (size >= 2ULL*1024*1024*1024) || byte_start || byte_end;
    hipFunction_t kernel = getMemcpyKernel(ctx, header_id, use64);
    hipStream_t cuStream = reinterpret_cast<hipStream_t>(stream);

    void* args[8];
    args[0] = &dst4;
    args[1] = &src4;
    args[2] = use64 ? (void*)&size_aligned : (void*)&size_aligned_32b;
    args[3] = &byte_start;
    args[4] = &byte_end;
    args[5] = &hash;
    args[6] = &sm_per_side;
    args[7] = &ctx.param_sm_side;

    hipModuleLaunchKernel(kernel, ctx.num_sms, 1, 1, 256, 4, 1, 0, cuStream, args, nullptr);
}

void sideaware_memcpy(void* dst, const void* src, size_t size, int device, hipStream_t stream) {
    sideaware_elementwise(dst, src, size, device, stream, 0);
}

// ---------------------------------------------------------------------------
// NVRTC: allow user to inject custom header code for future elementwise ops
// ---------------------------------------------------------------------------
int sideaware_set_custom_header(const char* header) {
    DeviceContext& ctx = get_device_context();

    if (!header || strlen(header)==0) {
        return 0; // default memcpy
    }

    std::string hdr_str(header);
    auto it = ctx.header_to_id.find(hdr_str);
    if (it != ctx.header_to_id.end()) {
        return it->second; // already assigned
    }

    int new_id = ctx.header_strings.size();
    ctx.header_strings.push_back(hdr_str);
    ctx.kernel_cache.emplace_back(); // default‑constructed
    ctx.header_to_id[hdr_str] = new_id;
    return new_id;
}

// ---------------------------------------------------------------------------
// Additional query/utility/config (mostly per device)
// ---------------------------------------------------------------------------
void fill_sm_sides_tensor(unsigned char* gpu_tensor) {
    DeviceContext& ctx = get_device_context();
    hipError_t err = hipMemcpy(gpu_tensor, ctx.gpu_side_index, ctx.num_sms, hipMemcpyDeviceToDevice);
    assert(err == hipSuccess);
}

const int* get_sm_side_summary_ptr()
{
    static int s[5];
    DeviceContext& ctx = get_device_context();

    s[0] = ctx.num_sms;
    s[1] = ctx.cpu_side_info[OFFSET_NUM_SM_SIDE0];
    s[2] = ctx.cpu_side_info[OFFSET_NUM_SM_SIDE1];
    s[3] = ctx.cpu_side_info[OFFSET_MIN_SM_PER_SIDE];
    s[4] = ctx.cpu_side_info[OFFSET_SIDE_HASH_MASK];
    return s;
}

void set_custom_alloc_threshold(size_t threshold) {
    g_custom_alloc_threshold = threshold;
}

void set_prealloc_config(int extra_required, int extra_alloc) {
    g_prealloc_extra_required = extra_required;
    g_prealloc_extra_alloc = extra_alloc;
}

void set_free_mapped_thresholds(size_t start_threshold, size_t end_threshold) {
    g_free_mapped_start_threshold = start_threshold;
    g_free_mapped_end_threshold = end_threshold;
}

void set_output_sass(const char* filename) {
    g_sass_filename = (filename && filename[0]) ? filename : "";
}

// ---------------------------------------------------------------------------
// Per-parameter query (rarely required)
// ---------------------------------------------------------------------------

int get_num_sms() {
    DeviceContext& ctx = get_device_context();
    return ctx.num_sms;
}

int get_num_sm_side0() {
    DeviceContext& ctx = get_device_context();
    return ctx.cpu_side_info[OFFSET_NUM_SM_SIDE0];
}

int get_num_sm_side1() {
    DeviceContext& ctx = get_device_context();
    return ctx.cpu_side_info[OFFSET_NUM_SM_SIDE1];
}

int get_min_sm_per_side() {
    DeviceContext& ctx = get_device_context();
    return ctx.cpu_side_info[OFFSET_MIN_SM_PER_SIDE];
}

int get_hash_mask() {
    DeviceContext& ctx = get_device_context();
    return ctx.cpu_side_info[OFFSET_SIDE_HASH_MASK];
}

} // extern "C"