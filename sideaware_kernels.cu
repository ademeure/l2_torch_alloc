// ---------------------------------------------------------------------------------
// Runtime‑compiled side‑aware memcpy kernel collection.
// ---------------------------------------------------------------------------------
#include <hip/hip_runtime.h>

// must match alloc.cu (TODO: dynamic)
constexpr unsigned int HASH = 0x2B3000;
constexpr unsigned int MAX_SM = 200;
constexpr unsigned int NUM_GROUPS = 4;
constexpr bool FORCE_WRONG_SIDE = false;
constexpr bool FORCE_RANDOM_SIDE = false;

constexpr unsigned int CHUNK_SIZE = 4096;

template<typename T, typename U>
struct is_same { static constexpr bool value = false; };
template<typename T>
struct is_same<T, T> { static constexpr bool value = true; };

typedef struct {
    unsigned char side_index[MAX_SM];
} param_sm_side_t;

struct unused {}; // type for inputs/outputs not used in the kernel

template<typename T, size_t N>
struct __align__(16) packed {
    T data[N];
    __device__ __forceinline__ T& operator[](int index) { return data[index]; }
    __device__ __forceinline__ const T& operator[](int index) const { return data[index]; }
};

template<bool evict = false, typename T, size_t N>
__device__ __forceinline__ packed<T, N> load(const packed<T, N> * __restrict__ src) {
    if constexpr (evict) {
        if constexpr (sizeof(T) * N == 16) {
            int4 data = __ldcs((const int4*)src);
            return *(packed<T, N>*)&data;
        } else if constexpr (sizeof(T) * N == 8) {
            int2 data = __ldcs((const int2*)src);
            return *(packed<T, N>*)&data;
        } else if constexpr (sizeof(T) * N == 4) {
            int data = __ldcs((const int*)src);
            return *(packed<T, N>*)&data;
        }
    }
    return *src;
}

typedef unsigned int o0;
typedef unused o1;
typedef unsigned int i0;
typedef unused i1;
typedef unused i2;
typedef unused i3;
constexpr bool input_evict[4] = {true, true, true, true};
constexpr bool input_discard[4] = {false, false, false, false};

constexpr bool reverse_order = true;
constexpr bool sideaware_for_o0 = false;
constexpr int parallel_iterations = 4;
constexpr int vec_size = 4;

typedef packed<o0, vec_size> vo0;
typedef packed<o1, vec_size> vo1;
typedef packed<i0, vec_size> vi0;
typedef packed<i1, vec_size> vi1;
typedef packed<i2, vec_size> vi2;
typedef packed<i3, vec_size> vi3;

__device__ __forceinline__ void discard_inputs(
    const vi0 * input0, const vi1 * input1, const vi2 * input2, const vi3 * input3) {
    if (input_discard[0] && !is_same<i0, unused>::value && (unsigned long long)input0 % 128 == 0)
        asm volatile("discard.global.L2 [%0], 128;\n" : : "l"(input0));
    if (input_discard[1] && !is_same<i1, unused>::value && (unsigned long long)input1 % 128 == 0)
        asm volatile("discard.global.L2 [%0], 128;\n" : : "l"(input1));
    if (input_discard[2] && !is_same<i2, unused>::value && (unsigned long long)input2 % 128 == 0)
        asm volatile("discard.global.L2 [%0], 128;\n" : : "l"(input2));
    if (input_discard[3] && !is_same<i3, unused>::value && (unsigned long long)input3 % 128 == 0)
        asm volatile("discard.global.L2 [%0], 128;\n" : : "l"(input3));
}

// ---------------------------------------------------------------------------------
// L2 Side Aware Multi-Input / Multi-Output Elementwise kernel (requires 16B alignment)
// ---------------------------------------------------------------------------------

template<typename size_type = long long>
__device__ __forceinline__ void elementwise_op(
        size_type element_idx, o0 &output0, o1 &output1,
        const i0 &input0, const i1 &input1, const i2 &input2, const i3 &input3) {
    // ...
    output0 = (o0)(input0);
    output1 = (o1)(input1);
}

template<typename size_type = long long>
__device__ __forceinline__ void vector_op(
        size_type vec_idx, vo0 &output0, vo1 &output1,
        const vi0 &input0, const vi1 &input1, const vi2 &input2, const vi3 &input3) {
    // ...
    for (int i = 0; i < vec_size; i++) {
        elementwise_op(vec_idx * vec_size + i, output0[i], output1[i], input0[i], input1[i], input2[i], input3[i]);
    }
}

template<typename size_type = long long>
__device__ __forceinline__ void side_aware_elementwise_device(
        vo0* __restrict__ output0, vo1* __restrict__ output1,
        const vi0 * __restrict__ input0, const vi1 * __restrict__ input1,
        const vi2 * __restrict__ input2, const vi3 * __restrict__ input3,
        size_type num_elements, unsigned int num_sm_per_side,
        const param_sm_side_t params) {
    // ...
    unsigned int smid;
    asm volatile("mov.u32 %0, %smid;\n" : "=r"(smid) :);

    unsigned int sm_side = params.side_index[smid] & 1;
    unsigned int sm_side_index = params.side_index[smid] >> 1;

    if (sm_side_index >= num_sm_per_side) {
        __nanosleep(1000);
        return;
    }

    // blockDim.x * element_size must equal CHUNK_SIZE (we check this on the host side)
    // e.g. 256 threads * 16 bytes = 4096 bytes
    constexpr size_type element_size = sideaware_for_o0 ? sizeof(vo0) : sizeof(vi0);

    int group = threadIdx.y;
    int group_tid_offset = threadIdx.x * element_size;
    int num_groups_per_side = num_sm_per_side * NUM_GROUPS;

    int num_double_chunks = (num_elements * element_size) / (2 * CHUNK_SIZE);
    int multi_chunks = num_double_chunks / parallel_iterations;

    size_type offset_per_group = (parallel_iterations * CHUNK_SIZE * 2);
    size_type offset_outer_loop = (offset_per_group * num_groups_per_side);
    offset_outer_loop = (reverse_order ? -offset_outer_loop : offset_outer_loop) - offset_per_group;

    int global_idx = (sm_side_index * NUM_GROUPS) + group;
    int adjusted_global_idx = reverse_order ? (multi_chunks - global_idx - 1) : global_idx;
    size_type byte_offset = adjusted_global_idx * offset_per_group + group_tid_offset;
    unsigned int base = (sideaware_for_o0 ? (unsigned long long)output0 : (unsigned long long)input0) & 0xFFFFFFFFULL;

    size_type elements[parallel_iterations];
    vi0 inputs0[parallel_iterations];
    vi1 inputs1[parallel_iterations];
    vi2 inputs2[parallel_iterations];
    vi3 inputs3[parallel_iterations];
    vo0 out0;
    vo1 out1;
    int j;

    #pragma unroll 1
    for (int i = global_idx; i < multi_chunks; i += num_groups_per_side, byte_offset += offset_outer_loop) {
        #pragma unroll parallel_iterations
        for (j = 0; j < parallel_iterations; j++, byte_offset += 2*CHUNK_SIZE) {
            unsigned int lsb_bits = base + (byte_offset & 0xFFFFFFFF);

            unsigned int side = __popc(lsb_bits & HASH) & 1;
            if constexpr (FORCE_WRONG_SIDE) side ^= 1;
            if constexpr (FORCE_RANDOM_SIDE) side = 0;

            unsigned int use_second_chunk = sm_side ^ side;
            size_type offset = byte_offset + (use_second_chunk * CHUNK_SIZE);

            size_type element = offset / element_size;
            elements[j] = element;
            inputs0[j] = load<input_evict[0]>(&input0[element]);
            inputs1[j] = load<input_evict[1]>(&input1[element]);
            inputs2[j] = load<input_evict[2]>(&input2[element]);
            inputs3[j] = load<input_evict[3]>(&input3[element]);
        }

        #pragma unroll parallel_iterations
        for (j = 0; j < parallel_iterations; j++) {
            size_type element = elements[j];
            discard_inputs(input0 + element, input1 + element, input2 + element, input3 + element);
        }

        #pragma unroll parallel_iterations
        for (j = 0; j < parallel_iterations; j++) {
            size_type element = elements[j];
            vector_op(element, out0, out1, inputs0[j], inputs1[j], inputs2[j], inputs3[j]);
            output0[element] = out0;
            output1[element] = out1;
        }
    }

    if (group == 0) {
        int max_remaining_double_chunks = (num_double_chunks + 1) - (multi_chunks * parallel_iterations);
        int start_sm_side_idx = num_sm_per_side - max_remaining_double_chunks;
        int idx = sm_side_index - start_sm_side_idx;
        if (idx >= 0) {
            size_type byte_offset = (size_type)(idx + multi_chunks*parallel_iterations) * (2*CHUNK_SIZE) + group_tid_offset;
            unsigned int lsb_bits = base + (byte_offset & 0xFFFFFFFF);
            unsigned int side = __popc(lsb_bits & HASH) & 1;

            unsigned int use_second_chunk = sm_side ^ side;
            byte_offset += use_second_chunk * CHUNK_SIZE;

            size_type element = byte_offset / element_size;
            if (element < num_elements) {
                j = 0;
                inputs0[j] = load<input_evict[0]>(input0 + element);
                inputs1[j] = load<input_evict[1]>(input1 + element);
                inputs2[j] = load<input_evict[2]>(input2 + element);
                inputs3[j] = load<input_evict[3]>(input3 + element);
                discard_inputs(input0 + element, input1 + element, input2 + element, input3 + element);

                vector_op(element, out0, out1, inputs0[j], inputs1[j], inputs2[j], inputs3[j]);
                output0[element] = out0;
                output1[element] = out1;
            }
        }
    }
}

// ---------------------------------------------------------------------------------
// Explicit wrapper kernels (external names – easier to locate with cuModuleGetFunction)
// ---------------------------------------------------------------------------------

extern "C" {

__global__ __launch_bounds__(1024, 1) void side_aware_memcpy_32(vo0* __restrict__ dst, const vi0* __restrict__ src,
        unsigned int num_elements, unsigned int sm_per_side, __grid_constant__ const param_sm_side_t params) {

    side_aware_elementwise_device<int>(dst, nullptr, src, nullptr, nullptr, nullptr,
                                                num_elements, sm_per_side, params);
}

__global__ __launch_bounds__(1024, 1) void side_aware_memcpy_64(vo0* __restrict__ dst, const vi0* __restrict__ src,
        size_t num_elements, unsigned int sm_per_side, __grid_constant__ const param_sm_side_t params) {

    side_aware_elementwise_device<long long>(dst, nullptr, src, nullptr, nullptr, nullptr,
                                          num_elements, sm_per_side, params);
}

} // extern "C"
