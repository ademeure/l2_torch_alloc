// ---------------------------------------------------------------------------------
// Runtime‑compiled side‑aware memcpy kernel collection.
// ---------------------------------------------------------------------------------
#include <hip/hip_runtime.h>

template<typename T, typename U>
struct is_same { static constexpr bool value = false; };
template<typename T>
struct is_same<T, T> { static constexpr bool value = true; };

constexpr int FORCED_UNROLL = 4; // TODO: make easy to configure
constexpr bool FORCE_WRONG_SIDE = false;
constexpr bool FORCE_RANDOM_SIDE = false;
constexpr unsigned int CHUNK_SIZE = 4096;

// must match alloc.cu (TODO: dynamic)
constexpr unsigned int MAX_SM = 200;
constexpr unsigned int NUM_GROUPS = 4;

typedef struct {
    unsigned char side_index[MAX_SM];
} param_sm_side_t;

struct unused {}; // type for inputs/outputs not used in the kernel

// ----------------------------------------------------------------------------
// L2 Side Aware memcpy kernel (single input, single output, any byte count)
// ----------------------------------------------------------------------------
template<typename size_type = size_t>
__device__ __forceinline__ void side_aware_memcpy_device(
        uint4 * __restrict__ output,
        const uint4 * __restrict__ input,
        size_type num_bytes, unsigned int hash, unsigned int num_sm_per_side,
        const param_sm_side_t params) {
    // ...
    unsigned int smid;
    asm volatile("mov.u32 %0, %smid;\n" : "=r"(smid) :);

    unsigned int sm_side = params.side_index[smid] & 1;
    unsigned int sm_side_index = params.side_index[smid] >> 1;

    if (sm_side_index >= num_sm_per_side) {
        __nanosleep(1000);
        return;
    }

    unsigned int group = threadIdx.y;
    unsigned int group_tid_offset = threadIdx.x * sizeof(uint4);
    unsigned int num_groups_per_side = num_sm_per_side * NUM_GROUPS;
    unsigned int global_idx = (sm_side_index * NUM_GROUPS) + group;

    unsigned int num_double_chunks = num_bytes / (2 * CHUNK_SIZE);
    unsigned int multi_chunks = num_double_chunks / FORCED_UNROLL;
    unsigned int base = ((unsigned long long)input) & 0xFFFFFFFFULL;

    unsigned int offset_per_group = (FORCED_UNROLL * CHUNK_SIZE * 2);
    unsigned int offset_outer_loop = (offset_per_group * num_groups_per_side) - offset_per_group;
    size_type byte_offset = global_idx * offset_per_group + group_tid_offset;

#pragma unroll
    for (unsigned int i = global_idx; i < multi_chunks; i += num_groups_per_side, byte_offset += offset_outer_loop) {
        size_type offsets[FORCED_UNROLL];
        uint4 inputs[FORCED_UNROLL];

        #pragma unroll FORCED_UNROLL
        for (int j = 0; j < FORCED_UNROLL; j++, byte_offset += 2*CHUNK_SIZE) {
            unsigned int lsb_bits = base + (byte_offset & 0xFFFFFFFF);
            unsigned int side = __popc(lsb_bits & hash) & 1;
            if constexpr (FORCE_WRONG_SIDE) side ^= 1;
            if constexpr (FORCE_RANDOM_SIDE) side = 0;

            unsigned int use_second_chunk = sm_side ^ side;
            size_type offset = byte_offset + (use_second_chunk * CHUNK_SIZE);

            offset /= sizeof(uint4);
            offsets[j] = offset;
            inputs[j] = input[offset];
        }
        #pragma unroll FORCED_UNROLL
        for (int j = 0; j < FORCED_UNROLL; j++) {
            output[offsets[j]] = inputs[j];
        }
    }

    if (group == 0) {
        int max_remaining_double_chunks = (num_double_chunks + 1) - (multi_chunks * FORCED_UNROLL);
        int start_sm_side_idx = num_sm_per_side - max_remaining_double_chunks;
        int idx = sm_side_index - start_sm_side_idx;
        if (idx >= 0) {
            size_type byte_offset = (size_type)(idx + multi_chunks*FORCED_UNROLL) * (2*CHUNK_SIZE) + group_tid_offset;
            unsigned int lsb_bits = base + (byte_offset & 0xFFFFFFFF);
            unsigned int side = __popc(lsb_bits & hash) & 1;

            unsigned int use_second_chunk = sm_side ^ side;
            byte_offset += use_second_chunk * CHUNK_SIZE;

            if (byte_offset + sizeof(uint4) <= num_bytes) {
                output[byte_offset / sizeof(uint4)] = input[byte_offset / sizeof(uint4)];
            }
        } else if (idx == -1 && sm_side == 0) {
            size_type byte_offset = threadIdx.x + num_bytes - (num_bytes % sizeof(uint4));
            if(byte_offset < num_bytes) {
                ((unsigned char*)output)[byte_offset] = ((unsigned char*)input)[byte_offset];
            }
        }
    }
}

// ---------------------------------------------------------------------------------
// Multi-Input / Multi-Output Elementwise kernel (generalised version of the above)
// ---------------------------------------------------------------------------------

template<typename size_type = size_t,
         typename o0 = uint4, typename o1 = uint4,
         typename i0 = uint4, typename i1 = uint4, typename i2 = uint4, typename i3 = uint4>
__device__ __forceinline__ void elementwise_op(
        size_type idx,
        o0 &output0, o1 &output1,
        const i0 &input0, const i1 &input1, const i2 &input2, const i3 &input3) {
    // ...
    output0 = input0;
    output1 = input1;
}

template<typename size_type = size_t, bool sideaware_output = false, bool aligned_16B = false,
        typename o0 = uint4, typename o1 = uint4,
        typename i0 = uint4, typename i1 = uint4, typename i2 = uint4, typename i3 = uint4>
__device__ __forceinline__ void side_aware_elementwise_device(
        o0* __restrict__ output0, o1* __restrict__ output1,
        const i0 * __restrict__ input0, const i1 * __restrict__ input1,
        const i2 * __restrict__ input2, const i3 * __restrict__ input3,
        size_type num_bytes, unsigned int byte_start, unsigned int byte_end,
        unsigned int hash, unsigned int num_sm_per_side,
        const param_sm_side_t params) {
    // ...
    unsigned int smid;
    asm volatile("mov.u32 %0, %smid;\n" : "=r"(smid) :);

    unsigned int sm_side = params.side_index[smid] & 1;
    unsigned int sm_side_index = params.side_index[smid] >> 1;

    if (sm_side_index >= num_sm_per_side) {
        __nanosleep(1000);
        return;
    }

    static_assert(sizeof(i0) == 16 || sizeof(o0) == 16, "i0 and/or o0 must be 16 bytes");
    constexpr size_type element_size = sideaware_output ? sizeof(o0) : sizeof(i0);
    size_type num_elements = num_bytes / element_size;

    unsigned int group = threadIdx.y;
    unsigned int group_tid_offset = threadIdx.x * sizeof(uint4);
    unsigned int num_groups_per_side = num_sm_per_side * NUM_GROUPS;
    unsigned int global_idx = (sm_side_index * NUM_GROUPS) + group;

    unsigned int num_double_chunks = num_bytes / (2 * CHUNK_SIZE);
    unsigned int multi_chunks = num_double_chunks / FORCED_UNROLL;
    unsigned int base = (sideaware_output ? (unsigned long long)output1 : (unsigned long long)input0) & 0xFFFFFFFFULL;

    unsigned int offset_per_group = (FORCED_UNROLL * CHUNK_SIZE * 2);
    unsigned int offset_outer_loop = (offset_per_group * num_groups_per_side) - offset_per_group;
    size_type byte_offset = global_idx * offset_per_group + group_tid_offset;

#pragma unroll
    for (unsigned int i = global_idx; i < multi_chunks; i += num_groups_per_side, byte_offset += offset_outer_loop) {
        size_type elements[FORCED_UNROLL];
        i0 inputs0[FORCED_UNROLL];
        i1 inputs1[FORCED_UNROLL];
        i2 inputs2[FORCED_UNROLL];
        i3 inputs3[FORCED_UNROLL];

        #pragma unroll FORCED_UNROLL
        for (int j = 0; j < FORCED_UNROLL; j++, byte_offset += 2*CHUNK_SIZE) {
            unsigned int lsb_bits = base + (byte_offset & 0xFFFFFFFF);

            unsigned int side = __popc(lsb_bits & hash) & 1;
            if constexpr (FORCE_WRONG_SIDE) side ^= 1;
            if constexpr (FORCE_RANDOM_SIDE) side = 0;

            unsigned int use_second_chunk = sm_side ^ side;
            size_type offset = byte_offset + (use_second_chunk * CHUNK_SIZE);

            size_type element = offset / element_size;
            elements[j] = element;
            inputs0[j] = input0[element];
            inputs1[j] = input1[element];
            inputs2[j] = input2[element];
            inputs3[j] = input3[element];
        }

        #pragma unroll FORCED_UNROLL
        for (int j = 0; j < FORCED_UNROLL; j++) {
            size_type element = elements[j];
            elementwise_op(element + (byte_start ? 1 : 0),
                           output0[element], output1[element],
                           inputs0[j], inputs1[j], inputs2[j], inputs3[j]);
        }
    }

    if (group == 0) {
        int max_remaining_double_chunks = (num_double_chunks + 1) - (multi_chunks * FORCED_UNROLL);
        int start_sm_side_idx = num_sm_per_side - max_remaining_double_chunks;
        int idx = sm_side_index - start_sm_side_idx;
        if (idx >= 0) {
            size_type byte_offset = (size_type)(idx + multi_chunks*FORCED_UNROLL) * (2*CHUNK_SIZE) + group_tid_offset;
            unsigned int lsb_bits = base + (byte_offset & 0xFFFFFFFF);
            unsigned int side = __popc(lsb_bits & hash) & 1;

            unsigned int use_second_chunk = sm_side ^ side;
            byte_offset += use_second_chunk * CHUNK_SIZE;

            size_type element = byte_offset / element_size;
            if (element < num_elements) {
                elementwise_op(element + (byte_start ? 1 : 0),
                               output0[element], output1[element],
                               input0[element], input1[element], input2[element], input3[element]);
            }
        } else if (aligned_16B == false && idx == -1 && sm_side == 0) {
            // Handle unaligned prefix (pre_bytes) and suffix (post_bytes) in a single pass.
            //   * Lane 0  deals with the trailing bytes of the first element.
            //   * Lane 32 deals with the leading bytes of the final element.
            // This avoids the previous negative / sentinel indices and copies only the
            // exact number of bytes that were skipped by the aligned main loop.
            o0 tmp0; o1 tmp1;
            unsigned char* bytes0 = reinterpret_cast<unsigned char*>(&tmp0);
            unsigned char* bytes1 = reinterpret_cast<unsigned char*>(&tmp1);

            /* Leading partial element -------------------------------------------------- */
            if (byte_start && threadIdx.x == 0) {
                elementwise_op(0, tmp0, tmp1, input0[-1], input1[-1], input2[-1], input3[-1]);

                unsigned char* out0 = reinterpret_cast<unsigned char*>(&output0[-1]);
                unsigned char* out1 = reinterpret_cast<unsigned char*>(&output1[-1]);

                const int byte_start_0 = (byte_start * sizeof(o0)) / element_size;
                const int byte_start_1 = (byte_start * sizeof(o1)) / element_size;

                #pragma unroll
                for (int b = 0; b < sizeof(uint4); b++) {
                    if (b >= byte_start_0 && b < sizeof(o0) && !is_same<o0, unused>::value) {
                        out0[b] = bytes0[b];
                    }
                    if (b >= byte_start_1 && b < sizeof(o1) && !is_same<o1, unused>::value) {
                        out1[b] = bytes1[b];
                    }
                }
            }

            /* Trailing partial element ------------------------------------------------- */
            if (byte_end && threadIdx.x == 32) {
                elementwise_op(num_elements + (byte_start ? 1 : 0), tmp0, tmp1,
                               input0[num_elements], input1[num_elements], input2[num_elements], input3[num_elements]);

                unsigned char* out0 = reinterpret_cast<unsigned char*>(&output0[num_elements]);
                unsigned char* out1 = reinterpret_cast<unsigned char*>(&output1[num_elements]);

                const int byte_end_0 = (byte_end * sizeof(o0)) / element_size;
                const int byte_end_1 = (byte_end * sizeof(o1)) / element_size;

                #pragma unroll
                for (int b = 0; b < sizeof(uint4); b++) {
                    if (b < byte_end_0 && b < sizeof(o0) && !is_same<o0, unused>::value) {
                        out0[b] = bytes0[b];
                    }
                    if (b < byte_end_1 && b < sizeof(o1) && !is_same<o1, unused>::value) {
                        out1[b] = bytes1[b];
                    }
                }
            }
        }
    }
}

// ---------------------------------------------------------------------------------
// Explicit wrapper kernels (external names – easier to locate with cuModuleGetFunction)
// ---------------------------------------------------------------------------------

extern "C" {
/*
__global__ void side_aware_elementwise_32(
        uint4* __restrict__ dst, const uint4* __restrict__ src,
        unsigned int num_bytes, unsigned int hash, unsigned int sm_per_side,
        __grid_constant__ const param_sm_side_t params) {

    side_aware_elementwise_device<unsigned int, false, uint4, unused, uint4, unused, unused, unused>(
        dst, nullptr, src, nullptr, nullptr, nullptr, num_bytes, hash, sm_per_side, params);
}
*/

__global__ __launch_bounds__(1024, 1) void side_aware_memcpy_32(
        uint4* __restrict__ dst, const uint4* __restrict__ src,
        unsigned int num_bytes_aligned, unsigned int byte_start, unsigned int byte_end,
        unsigned int hash, unsigned int sm_per_side,
        __grid_constant__ const param_sm_side_t params) {

    side_aware_elementwise_device<unsigned int, false, true, uint4, unused, uint4, unused, unused, unused>(
            dst, nullptr, src, nullptr, nullptr, nullptr,
            num_bytes_aligned, 0, 0, hash, sm_per_side, params);
}

__global__ __launch_bounds__(1024, 1) void side_aware_memcpy_64(
        uint4* __restrict__ dst, const uint4* __restrict__ src,
        size_t num_bytes_aligned, unsigned int byte_start, unsigned int byte_end,
        unsigned int hash, unsigned int sm_per_side,
        __grid_constant__ const param_sm_side_t params) {

    side_aware_elementwise_device<size_t, false, false, uint4, unused, uint4, unused, unused, unused>(
            dst, nullptr, src, nullptr, nullptr, nullptr,
            num_bytes_aligned, byte_start, byte_end, hash, sm_per_side, params);
}

} // extern "C"
